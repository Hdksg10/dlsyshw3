#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <sstream>

namespace needle {
namespace cuda {

#define BASE_THREAD_NUM 256
#define BASE_THREAD_NUM_2D 16
#define BLOCK_TILE_COL 1
#define TILE 4
#define S (BASE_THREAD_NUM_2D * TILE)
#define L (BLOCK_TILE_COL * TILE)
typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);

struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

CudaDims CudaTwoDim(size_t size_x, size_t size_y) {
  CudaDims dim;
  size_t num_blocks_x = (size_x + BASE_THREAD_NUM_2D - 1) / BASE_THREAD_NUM_2D;
  size_t num_blocks_y = (size_y + BASE_THREAD_NUM_2D - 1) / BASE_THREAD_NUM_2D;
  dim.block = dim3(BASE_THREAD_NUM_2D, BASE_THREAD_NUM_2D, 1);
  dim.grid = dim3(num_blocks_x, num_blocks_y, 1);
  return dim;
}

#define MAX_VEC_SIZE 8
struct CudaVec {
  uint32_t size;
  int32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<int32_t>& x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) throw std::runtime_error("Exceeded CUDA supported max dimesions");
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

////////////////////////////////////////////////////////////////////////////////
// Fill call
////////////////////////////////////////////////////////////////////////////////

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Compact and setitem cals
////////////////////////////////////////////////////////////////////////////////

// Untility function to convert contiguous index i to memory location from strides

__device__ static CudaVec GIDToIndices(size_t gid, CudaVec shape) {
  CudaVec indices;
  indices.size = shape.size;
  for (int i = indices.size - 1; i >= 0; i--) {
    indices.data[i] = gid % shape.data[i];
    gid = gid / shape.data[i];
  }  
  return indices;
}

__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the compact opeation.  This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  /// BEGIN SOLUTION
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gid < size) {
    size_t item_offset = offset;
    // convert gid to index array
    CudaVec indices = GIDToIndices(gid, shape);
    for (int i = 0; i < strides.size; i++) {
      item_offset = item_offset + indices.data[i] * strides.data[i];
    }
    // array copy
    out[gid] = a[item_offset];
  }
  // assert(false && "Not Implemented");
  /// END SOLUTION
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
             std::vector<int32_t> strides, size_t offset) {
  /**
   * Compact an array in memory.  Unlike the C++ version, in CUDA this will primarily call the 
   * relevant CUDA kernel.  In this case, we illustrate how you should set this up (i.e., we give 
   * you the code for this fuction, and also the prototype for the CompactKernel() function).  For
   * the functions after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset, being compact)
   */

  // Nothing needs to be added here
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
}

__global__ void EwiseSetitemKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    size_t item_offset = offset;
    // convert gid to index array
    CudaVec indices = GIDToIndices(gid, shape);
    for (int i = 0; i < strides.size; i++) {
      item_offset = item_offset + indices.data[i] * strides.data[i];
    }
    // array copy
    out[item_offset] = a[gid];
  }
  }

void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
                  std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items in a (non-compact) array using CUDA.  Yyou will most likely want to implement a
   * EwiseSetitemKernel() function, similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset, being compact)
   */
  /// BEGIN SOLUTION
  // assert(false && "Not Implemented");
  CudaDims dim = CudaOneDim(a.size);
  EwiseSetitemKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, a.size, VecToCuda(shape), VecToCuda(strides), offset);
  /// END SOLUTION
}

__global__ void ScalarSetitemKernel(scalar_t val, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    size_t item_offset = offset;
    // convert gid to index array
    CudaVec indices = GIDToIndices(gid, shape);
    for (int i = 0; i < strides.size; i++) {
      item_offset = item_offset + indices.data[i] * strides.data[i];
    }
    // array copy
    out[item_offset] = val;
  }
  }

void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<int32_t> shape,
                   std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  /// BEGIN SOLUTION
  // assert(false && "Not Implemented");
  CudaDims dim = CudaOneDim(size);
  ScalarSetitemKernel<<<dim.grid, dim.block>>>(val, out->ptr, size, VecToCuda(shape), VecToCuda(strides), offset);
  /// END SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

__global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + b[gid];
}

void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + val;
}

void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add together a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

/**
 * In the code the follows, use the above template to create analogous elementise
 * and and scalar operators for the following functions.  See the numpy backend for
 * examples of how they should work.
 *   - EwiseMul, ScalarMul
 *   - EwiseDiv, ScalarDiv
 *   - ScalarPower
 *   - EwiseMaximum, ScalarMaximum
 *   - EwiseEq, ScalarEq
 *   - EwiseGe, ScalarGe
 *   - EwiseLog
 *   - EwiseExp
 *   - EwiseTanh
 *
 * If you implement all these naively, there will be a lot of repeated code, so
 * you are welcome (but not required), to use macros or templates to define these
 * functions (however you want to do so, as long as the functions match the proper)
 * signatures above.
 */
#ifndef OPERATIONS
#define OPERATIONS

#define BINARY_EWISE_OP(name, op) \
__global__ void name##Kernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) { \
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
  if (gid < size) out[gid] = op(a[gid], b[gid]); \
}
#define BINARY_SCALAR_OP(name, op) \
__global__ void name##Kernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) { \
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
  if (gid < size) out[gid] = op(a[gid], val); \
}
#define UNARY_OP(name, op) \
__global__ void name##Kernel(const scalar_t* a, scalar_t* out, size_t size) { \
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
  if (gid < size) out[gid] = op(a[gid]); \
}

// Binary operations
#define MulOp(a, b) (a * b)
#define DivOp(a, b) (a / b)
#define MaxOp(a, b) (a > b ? a : b)
#define EqOp(a, b) (std::fabs(a - b) < 1e-6)
#define GeOp(a, b) (a >= b)
#define PowOp(a, b) std::pow(a, b)

// Unary operations
#define LogOp(a) std::log(a)
#define ExpOp(a) std::exp(a)
#define TanhOp(a) std::tanh(a)

BINARY_EWISE_OP(EwiseMul, MulOp)
BINARY_EWISE_OP(EwiseDiv, DivOp)
BINARY_EWISE_OP(EwiseMaximum, MaxOp)
BINARY_EWISE_OP(EwiseEq, EqOp)
BINARY_EWISE_OP(EwiseGe, GeOp)

BINARY_SCALAR_OP(ScalarMul, MulOp)
BINARY_SCALAR_OP(ScalarDiv, DivOp)
BINARY_SCALAR_OP(ScalarPower, PowOp)
BINARY_SCALAR_OP(ScalarMaximum, MaxOp)
BINARY_SCALAR_OP(ScalarEq, EqOp)
BINARY_SCALAR_OP(ScalarGe, GeOp)

UNARY_OP(EwiseLog, LogOp)
UNARY_OP(EwiseExp, ExpOp)
UNARY_OP(EwiseTanh, TanhOp)
#endif

void EwiseMul(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseMulKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

void EwiseDiv(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseDivKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

void EwiseMaximum(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

void EwiseEq(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseEqKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

void EwiseGe(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseGeKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

void ScalarMul(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarMulKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

void ScalarDiv(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarDivKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

void ScalarPower(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarPowerKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

void ScalarMaximum(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

void ScalarEq(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarEqKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

void ScalarGe(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarGeKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

void EwiseLog(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseLogKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

void EwiseExp(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseExpKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

void EwiseTanh(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseTanhKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

__global__ void MatmulKernel(const scalar_t *A, const scalar_t *B, scalar_t *out, uint32_t M, uint32_t N, uint32_t P) {
    // buffer in shared memory
    __shared__ float sA[S][L];
    __shared__ float sB[S][L]; // transpose shared buffer of matrix B
    // buffer in local memory
    float c[TILE][TILE] = {0};
    float a[TILE], b[TILE]; 
    size_t block_x = blockIdx.x;
    size_t block_y = blockIdx.y;
    for (int k = 0; k < N; k += L) {
        __syncthreads();
        // cooperative fetching
        int nthreads = blockDim.x * blockDim.y;
        int tid = blockDim.x * threadIdx.y + threadIdx.x;
        size_t item_id = tid; 
        while (item_id < S * L) {
            int x = item_id / L;
            int y = item_id % L;
            sA[x][y] = *(A + (block_x * S + x) * N + (k + y));
            sB[x][y] = *(B + (y + k) * P + (block_y * S + x));
          item_id += nthreads;
        }
        __syncthreads();
        
        for (int ki = 0; ki < L; ki++) {
            // copy to thread local vector
            for (int j = 0; j < TILE; j++) a[j] = sA[threadIdx.x * TILE + j][ki];
            for (int j = 0; j < TILE; j++) b[j] = sB[threadIdx.y * TILE + j][ki];
            // outer dots
            for (int x = 0; x < TILE; x++) {
                for (int y = 0; y < TILE; y++) {
                    c[x][y] += a[x] * b[y];
                }
            }
        }
    }

    size_t x_base = block_x * blockDim.x * TILE + threadIdx.x * TILE;
    size_t y_base = block_y * blockDim.y * TILE + threadIdx.y * TILE;

    // copy to out matrix
    for (int x = 0; x < TILE; x++) {
        for (int y = 0; y < TILE; y++) {
          if (x_base + x < M && y_base + y < P)
            *(out + (x_base + x) * P + (y_base + y)) = c[x][y];
        }
    }
}

void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out, uint32_t M, uint32_t N,
            uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling, 
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   * 
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */

  /// BEGIN SOLUTION
  // assert(false && "Not Implemented");
  CudaDims dim = CudaTwoDim(M, P);
  MatmulKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
  /// END SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////

__global__ void ReduceMaxKernel(const scalar_t* a, scalar_t* out, size_t size, size_t reduce_size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    size_t start = gid * reduce_size;
    scalar_t max_val = a[start];
    for (size_t i = 1; i < reduce_size; i++) {
      max_val = max_val > a[start + i] ? max_val : a[start + i];
    }
    out[gid] = max_val;
  }
}
__global__ void ReduceSumKernel(const scalar_t* a, scalar_t* out, size_t size, size_t reduce_size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    size_t start = gid * reduce_size;
    scalar_t sum_val = 0;
    for (size_t i = 0; i < reduce_size; i++) {
      sum_val = sum_val + a[start + i];
    }
    out[gid] = sum_val;
  }
}

void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.  Even though it is inefficient,
   * for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN SOLUTION
  // assert(false && "Not Implemented");
  CudaDims dim = CudaOneDim(out->size);
  ReduceMaxKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, reduce_size);
  /// END SOLUTION
}



void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.  Again, for simplicity you 
   * can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ReduceSumKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, reduce_size);
  /// END SOLUTION
}

}  // namespace cuda
}  // namespace needle

PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace needle;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from CPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides,
                       size_t offset) {
    std::vector<size_t> numpy_strides = strides;
    std::transform(numpy_strides.begin(), numpy_strides.end(), numpy_strides.begin(),
                   [](size_t& c) { return c * ELEM_SIZE; });

    // copy memory to host
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) throw std::bad_alloc();
    hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });

  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err =
        hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });

  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  m.def("ewise_mul", EwiseMul);
  m.def("scalar_mul", ScalarMul);
  m.def("ewise_div", EwiseDiv);
  m.def("scalar_div", ScalarDiv);
  m.def("scalar_power", ScalarPower);

  m.def("ewise_maximum", EwiseMaximum);
  m.def("scalar_maximum", ScalarMaximum);
  m.def("ewise_eq", EwiseEq);
  m.def("scalar_eq", ScalarEq);
  m.def("ewise_ge", EwiseGe);
  m.def("scalar_ge", ScalarGe);

  m.def("ewise_log", EwiseLog);
  m.def("ewise_exp", EwiseExp);
  m.def("ewise_tanh", EwiseTanh);

  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);
}
